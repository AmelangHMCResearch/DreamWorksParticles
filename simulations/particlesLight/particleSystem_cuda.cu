#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"
#include "event_timer.h"

extern "C"
{
    void cudaGLInit(int argc, char **argv)
    {
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        findCudaGLDevice(argc, (const char **)argv);
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsNone));
    }

    void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
    }

    void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
    {
        void *ptr;
        checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
                                                             *cuda_vbo_resource));
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
    }

    void copyArrayFromDevice(void *host, const void *device,
                             struct hipGraphicsResource **cuda_vbo_resource, int size)
    {
        if (cuda_vbo_resource)
        {
            device = mapGLBufferObject(cuda_vbo_resource);
        }

        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

        if (cuda_vbo_resource)
        {
            unmapGLBufferObject(*cuda_vbo_resource);
        }
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void integrateSystem(float *pos,
                         float *vel,
                         float *force,
                         float *posAfterLastSort,
                         float deltaTime,
                         uint numParticles,
                         bool posAfterLastSortIsValid,
                         bool *pointHasMovedMoreThanThreshold,
                         EventTimer* timer)                 
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        timer->startTimer(0, false);
        integrateSystemD<<< numBlocks, numThreads >>>((float4 *) pos,
                                                      (float4 *) vel,
                                                      (float4 *) force,
                                                      (float4 *) posAfterLastSort, 
                                                      deltaTime,
                                                      numParticles, 
                                                      posAfterLastSortIsValid, 
                                                      pointHasMovedMoreThanThreshold);
        timer->stopTimer(0, false);
    }

    void calcCellIndices(uint  *cellIndex,
                         uint  *particleIndex,
                         float *pos,
                         int    numParticles,
                         EventTimer* timer)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        timer->startTimer(1, true);
        calcCellIndicesD<<< numBlocks, numThreads >>>(cellIndex,
                                               particleIndex,
                                               (float4 *) pos,
                                               numParticles);
        timer->stopTimer(1, true);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }
    void sortParticles(uint *cellIndex, 
                       uint *particleIndex, 
                       uint numParticles, 
                       EventTimer* timer)
    {
        timer->startTimer(2, false);
        thrust::sort_by_key(thrust::device_ptr<uint>(cellIndex),
                            thrust::device_ptr<uint>(cellIndex + numParticles),
                            thrust::device_ptr<uint>(particleIndex));
        timer->stopTimer(2, false);
    }

    void sortParticlesOnce(uint *cellIndex, 
                       float *pos,
                       float *vel, 
                       uint numParticles, 
                       EventTimer* timer)
    {
        timer->startTimer(2, false);
        thrust::device_ptr<float4> pos4((float4 *)pos);
        thrust::device_ptr<float4> vel4((float4 *)vel);
        thrust::sort_by_key(thrust::device_ptr<uint>(cellIndex),
                            thrust::device_ptr<uint>(cellIndex + numParticles),
                            thrust::make_zip_iterator(thrust::make_tuple(pos4, vel4)));
        timer->stopTimer(2, false);
    }

    void copyArrays(float *pos,
                    float *tempPos,
                    float *vel,
                    float *tempVel,
                    uint   numParticles,
                    EventTimer* timer)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

#if USE_TEX
        checkCudaErrors(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
#endif

        timer->startTimer(3, true);
        copyArraysD<<< numBlocks, numThreads>>>(
            (float4 *)pos,
            (float4 *)tempPos,
            (float4 *)vel,
            (float4 *)tempVel,
            numParticles);
        timer->stopTimer(3, true);

#if USE_TEX
        checkCudaErrors(hipUnbindTexture(posTex));
        checkCudaErrors(hipUnbindTexture(velTex));
#endif

    }

    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     uint  *cellIndex,
                                     uint  *particleIndex,
                                     float *pos,
                                     float *tempPos,
                                     float *posAfterLastSort,
                                     float *vel,
                                     float *tempVel,
                                     bool  *posAfterLastSortIsValid,
                                     bool  *pointHasMovedMoreThanThreshold, 
                                     uint   numParticles,
                                     uint   numCells,
                                     EventTimer* timer)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        uint smemSize = sizeof(uint)*(numThreads+1);

#if USE_TEX
        checkCudaErrors(hipBindTexture(0, tempPosTex, tempPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, tempVelTex, tempVel, numParticles*sizeof(float4)));
#endif

        timer->startTimer(3, true);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
            cellStart,
            cellEnd,
            cellIndex,
            particleIndex,
            (float4 *) pos,
            (float4 *) tempPos,
            (float4 *) posAfterLastSort,
            (float4 *) vel,
            (float4 *) tempVel,
            pointHasMovedMoreThanThreshold,
            numParticles);
        timer->stopTimer(3, true);
        *posAfterLastSortIsValid = true;
#if USE_TEX
        checkCudaErrors(hipUnbindTexture(tempPosTex));
        checkCudaErrors(hipUnbindTexture(tempVelTex));
#endif

        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");
    }

    void findCellStart(uint  *cellStart,
                       uint  *cellEnd,
                       uint  *cellIndex,
                       float *pos,
                       float *oldPos,
                       uint   numParticles,
                       uint   numCells,
                       EventTimer* timer)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));


#if USE_TEX
        checkCudaErrors(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
#endif

        uint smemSize = sizeof(uint)*(numThreads+1);
        timer->startTimer(3, true);
        findCellStartD<<< numBlocks, numThreads, smemSize>>>(
            cellStart,
            cellEnd,
            cellIndex,
            (float4 *)pos,
            (float4 *)oldPos,
            numParticles);
        timer->stopTimer(3, true);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
        checkCudaErrors(hipUnbindTexture(posTex));
#endif
    }

    void collide(float *pos,
                 float *vel,
                 float *force,
                 uint  *cellIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint  *numNeighbors,
                 uint   numParticles,
                 uint   numCells,
                 EventTimer* timer)
    {
#if USE_TEX
        checkCudaErrors(hipBindTexture(0, posTex, pos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, velTex, vel, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
#endif

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        timer->startTimer(4, true);
        collideD<<< numBlocks, numThreads >>>((float4 *)pos,
                                              (float4 *)vel,
                                              (float4 *)force,
                                              cellIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles,
                                              numNeighbors);
        timer->stopTimer(4, true);
    
        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

#if USE_TEX
        checkCudaErrors(hipUnbindTexture(posTex));
        checkCudaErrors(hipUnbindTexture(velTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
#endif
    }

    bool checkForResort(bool *pointHasMovedMoreThanThreshold)
    {
      bool needsResort;
      hipMemcpy(&needsResort, pointHasMovedMoreThanThreshold, sizeof(bool), hipMemcpyDeviceToHost);
      return needsResort;
    }

}   // extern "C"
