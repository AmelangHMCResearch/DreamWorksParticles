#include "hip/hip_runtime.h"


#include <math.h>

#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime_api.h>

#include "gpuVoxelTree.h"

__constant__ unsigned int numLevels;
__constant__ BoundingBox  boundary;
__constant__ unsigned int  numCellsPerSide[10];
__constant__ float voxelSize;
__constant__ float* pointersToStatuses[10];
__constant__ unsigned int* pointersToDelimiters[10]; // Don't need delimiters for the lowest level
__constant__ unsigned int voxelsPerSide; 

// textures for particle position and velocity
//texture<uint, 1, hipReadModeElementType> voxelStrengthTex;
texture<uint, 1, hipReadModeElementType> triTex;
texture<uint, 1, hipReadModeElementType> numVertsTex;


// Utility Functions
void getPointersToDeallocateFromGPU(std::vector<void *> statusPointersToDeallocate, 
                                    std::vector<void *> delimiterPointersToDeallocate,
                                    uint numLevels)
{
	checkCudaErrors(hipMemcpyFromSymbol(&statusPointersToDeallocate[0], HIP_SYMBOL(pointersToStatuses),
                                         numLevels * sizeof(float *), 0, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpyFromSymbol(&delimiterPointersToDeallocate[0], HIP_SYMBOL(pointersToDelimiters),
                                         numLevels * sizeof(unsigned int *), 0, hipMemcpyDeviceToHost));
}
void copyDataToConstantMemory(unsigned int numberOfLevels,
                             BoundingBox BB, 
                             std::vector<unsigned int> numberOfCellsPerSide,
                             float sizeOfVoxel,
                             std::vector<void *> pointersToLevelStatuses,
                             std::vector<void *> pointersToLevelDelimiters,
                             unsigned int numberOfVoxelsPerSide)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(numLevels), (void *) &numberOfLevels, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(boundary), (void *) &BB, sizeof(BoundingBox)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(numCellsPerSide), (void *) &numberOfCellsPerSide[0], numberOfLevels * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxelSize), (void *) &sizeOfVoxel, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pointersToStatuses), (void *) &pointersToLevelStatuses[0], numberOfLevels * sizeof(float *)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pointersToDelimiters), (void *) &pointersToLevelDelimiters[0], numberOfLevels * sizeof(unsigned int *)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxelsPerSide), (void *) &numberOfVoxelsPerSide, sizeof(unsigned int)));
}

// Functions for CUDA
__device__
uint3 calculateCoordsFromIndex(uint index)
{
    // Find the coordinates of a *marching cube* from its index
    uint3 center;
    center.z = index / ((voxelsPerSide + 1) * (voxelsPerSide + 1));
    center.y = (index - center.z * (voxelsPerSide + 1) * (voxelsPerSide + 1)) / (voxelsPerSide + 1); 
    center.x = index - (voxelsPerSide + 1) * (center.y + (voxelsPerSide + 1) * center.z);
    return center;
}


__device__
float3 calculateVoxelCenter(int3 gridPos)
{
    float3 center;
    center.x = boundary.lowerBoundary.x + (voxelSize / 2.0) + gridPos.x * voxelSize;
    center.y = boundary.lowerBoundary.x + (voxelSize / 2.0) + gridPos.y * voxelSize;
    center.z = boundary.lowerBoundary.x + (voxelSize / 2.0) + gridPos.z * voxelSize;
    return center;
}

__device__
float tangle(float x, float y, float z)
{
    x *= 3.0f;
    y *= 3.0f;
    z *= 3.0f;
    return (x*x*x*x - 5.0f*x*x +y*y*y*y - 5.0f*y*y +z*z*z*z - 5.0f*z*z + 11.8f) * 0.2f + 0.5f;
}

// evaluate field function at a point
// returns value and gradient in float4
__device__
float4 fieldFunc4(float3 p)
{
    float v = tangle(p.x, p.y, p.z);
    const float d = 0.001f;
    float dx = tangle(p.x + d, p.y, p.z) - v;
    float dy = tangle(p.x, p.y + d, p.z) - v;
    float dz = tangle(p.x, p.y, p.z + d) - v;
    return make_float4(dx, dy, dz, v);
}

__device__
void vertexInterp2(float isolevel, float3 p0, float3 p1, float4 f0, float4 f1, float3 &p, float3 &n)
{
    float t = (isolevel - f0.w) / (f1.w - f0.w);
    p = lerp(p0, p1, 0.5);
    n.x = lerp(f0.x, f1.x, t);
    n.y = lerp(f0.y, f1.y, t);
    n.z = lerp(f0.z, f1.z, t);
    n = normalize(n);
}

__device__
unsigned int getCell(float3 pos, BoundingBox boundingBox, unsigned int cubeSize)
{
    // "origin" of box is at the lower boundary
	float3 relPos = pos + -1.0 * boundingBox.lowerBoundary; 
    float sizeOfCell = (boundingBox.upperBoundary.x - boundingBox.lowerBoundary.x) / (float) cubeSize; 
    // Find which cell the position is in
    uint xCoord = (uint) floor(relPos.x / sizeOfCell); 
    uint yCoord = (uint) floor(relPos.y / sizeOfCell); 
    uint zCoord = (uint) floor(relPos.z / sizeOfCell); 
    return zCoord * cubeSize * cubeSize + yCoord * cubeSize + xCoord; 
}

__device__
BoundingBox calculateNewBoundingBox(float3 pos, BoundingBox boundingBox, uint cubeSize)
{
    // Find which cell of the old bounding box the pos is in
	float3 offsetFromOrigin = pos + (-1.0f * boundingBox.lowerBoundary);
    float sizeOfCell = (boundingBox.upperBoundary.x - boundingBox.lowerBoundary.x) / (float) cubeSize; 
	uint3 lowerIndex;
	lowerIndex.x = (uint) floor(offsetFromOrigin.x / sizeOfCell);
	lowerIndex.y = (uint) floor(offsetFromOrigin.y / sizeOfCell);
	lowerIndex.z = (uint) floor(offsetFromOrigin.z / sizeOfCell);
    // Calculate the new upper and lower boundaries based on the cell
	BoundingBox newBB; 
    newBB.lowerBoundary = make_float3(lowerIndex) * sizeOfCell; 
    newBB.upperBoundary = make_float3((lowerIndex + make_uint3(1,1,1))) * sizeOfCell; 
    return newBB; 
}

__device__
bool isOutsideBoundingBox(float3 pos)
{
    float3 lowerBound = boundary.lowerBoundary;
    float3 upperBound = boundary.upperBoundary; 
    if (pos.x < lowerBound.x || pos.y < lowerBound.y || pos.z < lowerBound.z) {
        return true; 
    }
    if (pos.x > upperBound.x || pos.y > upperBound.y || pos.z > upperBound.z) {
        return true;
    }
    return false; 
}


__device__
unsigned int getStatus(float3 pos)
{
    // Start at level 0, offset into cell 0, and the bounding box for the whole gdb
	unsigned int currentLevel = 0;
	BoundingBox currentBB = boundary;
	unsigned int offset = 0; 
    if (isOutsideBoundingBox(pos)) {
        // If outside the bounding box, the voxel is inactive
        return 0.0; 
    }
	while (1) {
        // Otherwise, get the status of the cell we're in
		unsigned int cell = getCell(pos, currentBB, numCellsPerSide[currentLevel]);
		float status = pointersToStatuses[currentLevel][cell + offset];
		// Dig deeper = INF
		if (status != NAN) {
            // If it is active or inactive, return the status
			return status;
		} else {
            // Otherwise, find our new offset and bounding box, and loop
			unsigned int delimiter = pointersToDelimiters[currentLevel][cell + offset];
			unsigned int nextLevelCubeSize = numCellsPerSide[currentLevel + 1];
			offset = delimiter * nextLevelCubeSize * nextLevelCubeSize * nextLevelCubeSize; 
			currentBB = calculateNewBoundingBox(pos, currentBB, numCellsPerSide[currentLevel + 1]);
		}

	}
}

__global__
void calculateNewVelocities(float4 *particlePos,
                            float4 *particleVel,
                            float particleRadius,
                            unsigned int numParticles,
                            float deltaTime, 
                            float4 *result,
                            unsigned int *sizeOfResult,
                            unsigned int maxResultSize)
{
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index >= numParticles) return;

    float3 currentParticlePos = make_float3(particlePos[index]);
    float3 currentParticleVel = make_float3(particleVel[index]);
    float iters = particleVel[index].w; 

    // Loop over all voxels that are touching the particle
    int loopStart = -1.0 * floor(particleRadius / voxelSize);
    int loopEnd = ceil(particleRadius / voxelSize); 

    float3 averagePosition = make_float3(0,0,0); 
    unsigned int numNeighboringVoxels = 0; 

    for (int z = loopStart; z <= loopEnd; ++z) {
    	for (int y = loopStart; y <= loopEnd; ++y) {
    		for (int x = loopStart; x <= loopEnd; ++x) {
    			float3 position = currentParticlePos + voxelSize * make_float3(x, y, z); 
    			float status = getStatus(position); 
    			if (status > 0) {
                    // Get data for average voxel position
    				++numNeighboringVoxels; 
                    averagePosition += position;
                    // Reduce the strength (do so more for glancing blows) 
                    float t_c = 0.1; 
                    float amountToReduceStrength = -10.0 * length(cross(currentParticleVel, currentParticlePos - position)) * deltaTime / t_c;
                    unsigned int indexToAdd = atomicAdd(sizeOfResult, 1);
                    if (indexToAdd < maxResultSize) {
                        // Add position of voxel and amount to reduce strength to our output for later use
                        result[indexToAdd] = make_float4(position, amountToReduceStrength);  
                    }  
    			}
    		}
    	}
    }

    if (numNeighboringVoxels > 0) {
        // get the average position
        averagePosition.x = averagePosition.x / numNeighboringVoxels;
        averagePosition.y = averagePosition.y / numNeighboringVoxels;
        averagePosition.z = averagePosition.z / numNeighboringVoxels;
        
        // The particle reflects around the normal.  
        float3 normalVector = (currentParticlePos - averagePosition) / length(currentParticlePos - averagePosition);
        currentParticleVel -= 2 * dot(normalVector, currentParticleVel) * normalVector;
        currentParticlePos = averagePosition + (2 * particleRadius * normalVector);

        // TODO: Figure out a way to remove particles
    }
    particlePos[index] = make_float4(currentParticlePos, 1.0f);
    particleVel[index] = make_float4(currentParticleVel, iters);
}

__global__
void repairVoxelTree(float4 *result,
                     unsigned int *numClaimedInArrayAtLevel,
                     unsigned int numToRepair)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index >= numToRepair) return;

    float WORK_IN_PROGRESS = INFINITY; 
    float3 pos = make_float3(result[index]); 
    BoundingBox currentBB = boundary; 
    //unsigned int cell; 
    unsigned int offset = 0;
    unsigned int numCellsInLevel = 1; 
    unsigned int cell;
    float amountToReduceStrength = result[index].w;

    for (int level = 0; level < numLevels - 1; ++level) {
        // Get index of cell 
        cell = getCell(pos, currentBB, numCellsPerSide[level]);
        numCellsInLevel *= numCellsPerSide[level] * numCellsPerSide[level] * numCellsPerSide[level]; 
        if (cell + offset >= numCellsInLevel) {
            printf("Problem1 at index %d\n", index);
            return; 
        }
        // Check if work is happening already
        float prevStatus = atomicExch(&(pointersToStatuses[level][cell + offset]), WORK_IN_PROGRESS);
        if (prevStatus != WORK_IN_PROGRESS) {
            // If chunk is not allready dig deeper...
            int chunkNum = pointersToDelimiters[level][cell + offset];
            if (chunkNum == -1) {
                // Get pos to add into
                unsigned int positionToAdd = atomicAdd(&numClaimedInArrayAtLevel[level + 1], 1);
                unsigned int numCellsInChunk = numCellsPerSide[level + 1] * numCellsPerSide[level + 1] * numCellsPerSide[level + 1];

                unsigned int numCellsInNextLevel = numCellsInLevel * numCellsPerSide[level + 1] * numCellsPerSide[level + 1] * numCellsPerSide[level + 1];
                for (int i = 0; i < numCellsInChunk; ++i) {
                    if (positionToAdd * numCellsInChunk + i >= numCellsInNextLevel) {
                       printf("Problem2 at index %d\n", index);
                       return; 
                    }
                    // Set next level to proper values
                    pointersToStatuses[level + 1][positionToAdd * numCellsInChunk + i] = 1;
                    pointersToDelimiters[level + 1][positionToAdd * numCellsInChunk + i] = -1; 
                }
                // Update chunk index
                pointersToDelimiters[level][cell + offset] = positionToAdd; 
            }
            //NAN = dig deeper into tree
            pointersToStatuses[level][cell + offset] = NAN; 
        } else {
            while (pointersToStatuses[level][cell + offset] == WORK_IN_PROGRESS) {}
        }
        unsigned int delimiter = pointersToDelimiters[level][cell + offset];
        unsigned int nextLevelCubeSize = numCellsPerSide[level + 1];
        offset = delimiter * nextLevelCubeSize * nextLevelCubeSize * nextLevelCubeSize; 
        currentBB = calculateNewBoundingBox(pos, currentBB, numCellsPerSide[level + 1]);
    }
    // TODO: Is amount to Reduce strength negative?
    atomicAdd(&pointersToStatuses[numLevels - 1][cell + offset], amountToReduceStrength);
    return; 	
}

__global__
void coarsenVoxelTree(float4 *result)
{
	return; 
}

void collideWithParticles(float *particlePos,
                          float *particleVel,
                          float  particleRadius,
                          unsigned int numParticles,
                          unsigned int *numClaimedInArrayAtLevel,
                          float deltaTime)
{
	unsigned int numThreads = 256; 
	unsigned int numBlocks = ceil((float) numParticles / numThreads);
    unsigned int maxResultSize = 10000;
	float *result;
    checkCudaErrors(hipMalloc((void **) &result, maxResultSize * sizeof(float4))); 
    unsigned int *sizeOfResult; 
    checkCudaErrors(hipMalloc((void **) &sizeOfResult, 1 * sizeof(unsigned int))); 
    checkCudaErrors(hipMemset(sizeOfResult, 0, sizeof(unsigned int))); 
    calculateNewVelocities<<<numBlocks, numThreads>>>((float4 *) particlePos,
                                                    (float4 *) particleVel,
                                                    particleRadius,
                                                    numParticles,
                                                    deltaTime,
                                                    (float4 *) result,
                                                    sizeOfResult, 
                                                    maxResultSize);
    getLastCudaError("Kernel execution failed");


    unsigned int size; 
    hipMemcpy(&size, sizeOfResult, sizeof(unsigned int), hipMemcpyDeviceToHost);
    numThreads = min(256, size); 
    if (numThreads > 0) {
        numBlocks = ceil((float) size / numThreads); 
        if (size > maxResultSize) 
            printf("problem: Size %d is greater than max size %d\n", size, maxResultSize);
        repairVoxelTree<<<numBlocks, numThreads>>>((float4 *) result,
                                                    numClaimedInArrayAtLevel, 
                                                    min(size, maxResultSize));
    }
    getLastCudaError("Kernel execution failed");

    //coarsenVoxelTree<<<numBlocks, numThreads>>>((float4 *) result);
    hipFree(result);
    hipFree(sizeOfResult); 
	
}

__global__
void createMarchingCubesMeshD(float4 *vertexPos,
                              float4 *norm,
                              uint  *tri,
                              uint  *numVerts,
                              uint  *numVerticesClaimed,
                              uint   numVoxelsToDraw)
{
    uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

    // Get gridPos of our grid cube - starts as lower left corner as (0,0,0)
    uint3 gridPos = calculateCoordsFromIndex(index);
    //printf("GridPos: %d, %d, %d numCells: %d voxelSize: %f\n", gridPos.x, gridPos.y, gridPos.z, voxelsPerSide, voxelSize);
    
    // Check if voxels on corner of gridcube are 
    int lookupIndexForActiveVertices = 0;
    float3 cubeVertexPos[8];
    float4 field[8];

    int3 i = make_int3(-1, -1, -1);
    int3 toCheck = make_int3(gridPos) + i;
    cubeVertexPos[0] = calculateVoxelCenter(toCheck);
    bool isActive = getStatus(cubeVertexPos[0]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 0); 
    field[0] = fieldFunc4(cubeVertexPos[0]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 0, lookupIndexForActiveVertices, cubeVertexPos[0].x, cubeVertexPos[0].y, cubeVertexPos[0].z);

    
    i = make_int3(0,-1,-1);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[1] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[1]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 1); 
    field[1] = fieldFunc4(cubeVertexPos[1]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 1, lookupIndexForActiveVertices, cubeVertexPos[1].x, cubeVertexPos[1].y, cubeVertexPos[1].z);

    i = make_int3(0,0,-1);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[2] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[2]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 2); 
    field[2] = fieldFunc4(cubeVertexPos[2]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 2, lookupIndexForActiveVertices, cubeVertexPos[2].x, cubeVertexPos[2].y, cubeVertexPos[2].z);

    i = make_int3(-1,0,-1);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[3] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[3]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 3); 
    field[3] = fieldFunc4(cubeVertexPos[3]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 3, lookupIndexForActiveVertices, cubeVertexPos[3].x, cubeVertexPos[3].y, cubeVertexPos[3].z);

    i = make_int3(-1,-1,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[4] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[4]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 4); 
    field[4] = fieldFunc4(cubeVertexPos[4]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 4, lookupIndexForActiveVertices, cubeVertexPos[4].x, cubeVertexPos[4].y, cubeVertexPos[4].z);

    i = make_int3(0,-1,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[5] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[5]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 5); 
    field[5] = fieldFunc4(cubeVertexPos[5]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 5, lookupIndexForActiveVertices, cubeVertexPos[5].x, cubeVertexPos[5].y, cubeVertexPos[5].z);

    i = make_int3(0,0,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[6] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[6]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 6); 
    field[6] = fieldFunc4(cubeVertexPos[6]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 6, lookupIndexForActiveVertices, cubeVertexPos[6].x, cubeVertexPos[6].y, cubeVertexPos[6].z);

    i = make_int3(-1,0,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[7] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[7]) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 7); 
    field[7] = fieldFunc4(cubeVertexPos[7]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 7, lookupIndexForActiveVertices, cubeVertexPos[7].x, cubeVertexPos[7].y, cubeVertexPos[7].z);
    
    float3 vertlist[12];
    float3 normList[12];

    vertexInterp2(0.0, cubeVertexPos[0], cubeVertexPos[1], field[0], field[1], vertlist[0], normList[0]);
    vertexInterp2(0.0, cubeVertexPos[1], cubeVertexPos[2], field[1], field[2], vertlist[1], normList[1]);
    vertexInterp2(0.0, cubeVertexPos[2], cubeVertexPos[3], field[2], field[3], vertlist[2], normList[2]);
    vertexInterp2(0.0, cubeVertexPos[3], cubeVertexPos[0], field[3], field[0], vertlist[3], normList[3]);

    vertexInterp2(0.0, cubeVertexPos[4], cubeVertexPos[5], field[4], field[5], vertlist[4], normList[4]);
    vertexInterp2(0.0, cubeVertexPos[5], cubeVertexPos[6], field[5], field[6], vertlist[5], normList[5]);
    vertexInterp2(0.0, cubeVertexPos[6], cubeVertexPos[7], field[6], field[7], vertlist[6], normList[6]);
    vertexInterp2(0.0, cubeVertexPos[7], cubeVertexPos[4], field[7], field[4], vertlist[7], normList[7]);

    vertexInterp2(0.0, cubeVertexPos[0], cubeVertexPos[4], field[0], field[4], vertlist[8], normList[8]);
    vertexInterp2(0.0, cubeVertexPos[1], cubeVertexPos[5], field[1], field[5], vertlist[9], normList[9]);
    vertexInterp2(0.0, cubeVertexPos[2], cubeVertexPos[6], field[2], field[6], vertlist[10], normList[10]);
    vertexInterp2(0.0, cubeVertexPos[3], cubeVertexPos[7], field[3], field[7], vertlist[11], normList[11]);

    uint numVerticesToAdd = tex1Dfetch(numVertsTex, lookupIndexForActiveVertices);
    uint positionToAdd = atomicAdd(numVerticesClaimed, numVerticesToAdd); 
    //if (gridPos.z ==0 && index % 1000 == 0) printf("To add: %d Pos: %d Total: %d\n", numVerticesToAdd, positionToAdd, numVoxelsToDraw * 15);
    for (int i= 0; i < numVerticesToAdd; ++i) {

        uint edge = tex1Dfetch(triTex, lookupIndexForActiveVertices*16 + i);
        uint indexToAdd = positionToAdd + i;

        if (indexToAdd < numVoxelsToDraw * 15)
        {
            vertexPos[indexToAdd] = make_float4(vertlist[edge], 1.0f);
            norm[indexToAdd] = make_float4(normList[edge], 0.0f);
        }
    }

}

void generateMarchingCubes(float *pos,
                           float *norm,
                           unsigned int *tri,
                           unsigned int *numVerts,
                           unsigned int *verticesInPosArray,
                           unsigned int numVoxelsToDraw,
                           unsigned int numMarchingCubes)
{
    checkCudaErrors(hipBindTexture(0, triTex, tri, sizeof(uint) * 256 * 16));
    checkCudaErrors(hipBindTexture(0, numVertsTex, numVerts, sizeof(uint) * 256));

    // thread per particle
    uint numThreads, numBlocks;
    numThreads = 256;
    numBlocks = ceil((float) numMarchingCubes / (float) numThreads);
    hipMemset(verticesInPosArray, 0, sizeof(uint));

    // execute the kernel
    //timer->startTimer(5, true);
    createMarchingCubesMeshD<<< numBlocks, numThreads >>>((float4 *) pos,
                                                          (float4 *) norm,
                                                           tri,
                                                           numVerts,
                                                           verticesInPosArray,
                                                           numVoxelsToDraw);
    //timer->stopTimer(5, true);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipUnbindTexture(triTex));
    checkCudaErrors(hipUnbindTexture(numVertsTex));
}