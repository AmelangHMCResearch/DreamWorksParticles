#include "hip/hip_runtime.h"


#include <math.h>
#include <limits.h>

#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime_api.h>

#include "gpuVoxelTree.h"

__constant__ unsigned int numLevels;
__constant__ BoundingBox  boundary;
__constant__ unsigned int  numCellsPerSide[10];
__constant__ float voxelSize;
//__constant__ float* pointersToStatuses[10];
//__constant__ unsigned int* pointersToDelimiters[10]; // Don't need delimiters for the lowest level
__constant__ unsigned int voxelsPerSide; 

// textures for particle position and velocity
//texture<uint, 1, hipReadModeElementType> voxelStrengthTex;
texture<uint, 1, hipReadModeElementType> triTex;
texture<uint, 1, hipReadModeElementType> numVertsTex;

// I don't like #defines, but we can't do static const variables because
//  they have to be available to host and device.  grrr...
#define STATUS_FLAG_WORK_IN_PROGRESS INFINITY
#define STATUS_FLAG_DIG_DEEPER (-1.0 * INFINITY)       

// Utility Functions
void getPointersToDeallocateFromGPU(std::vector<void *> statusPointersToDeallocate, 
                                    std::vector<void *> delimiterPointersToDeallocate,
                                    uint numLevels)
{
	/*checkCudaErrors(hipMemcpyFromSymbol(&statusPointersToDeallocate[0], HIP_SYMBOL(pointersToStatuses),
                                         numLevels * sizeof(float *), 0, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpyFromSymbol(&delimiterPointersToDeallocate[0], HIP_SYMBOL(pointersToDelimiters),
                                         numLevels * sizeof(unsigned int *), 0, hipMemcpyDeviceToHost));*/
}
void copyDataToConstantMemory(unsigned int numberOfLevels,
                             BoundingBox BB, 
                             std::vector<unsigned int> numberOfCellsPerSide,
                             float sizeOfVoxel,
                             std::vector<void *> pointersToLevelStatuses,
                             std::vector<void *> pointersToLevelDelimiters,
                             unsigned int numberOfVoxelsPerSide)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(numLevels), (void *) &numberOfLevels, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(boundary), (void *) &BB, sizeof(BoundingBox)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(numCellsPerSide), (void *) &numberOfCellsPerSide[0], numberOfLevels * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxelSize), (void *) &sizeOfVoxel, sizeof(float)));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pointersToStatuses), (void *) &pointersToLevelStatuses[0], numberOfLevels * sizeof(float *)));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pointersToDelimiters), (void *) &pointersToLevelDelimiters[0], numberOfLevels * sizeof(unsigned int *)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxelsPerSide), (void *) &numberOfVoxelsPerSide, sizeof(unsigned int)));
}

// Functions for CUDA
__device__
uint3 calculateCoordsFromIndex(uint index)
{
    // Find the coordinates of a *marching cube* from its index
    uint3 center;
    center.z = index / ((voxelsPerSide + 1) * (voxelsPerSide + 1));
    center.y = (index - center.z * (voxelsPerSide + 1) * (voxelsPerSide + 1)) / (voxelsPerSide + 1); 
    center.x = index - (voxelsPerSide + 1) * (center.y + (voxelsPerSide + 1) * center.z);
    return center;
}


__device__
float3 calculateVoxelCenter(int3 gridPos)
{
    float3 center;
    center.x = boundary.lowerBoundary.x + (voxelSize / 2.0) + gridPos.x * voxelSize;
    center.y = boundary.lowerBoundary.x + (voxelSize / 2.0) + gridPos.y * voxelSize;
    center.z = boundary.lowerBoundary.x + (voxelSize / 2.0) + gridPos.z * voxelSize;
    return center;
}

__device__
float tangle(float x, float y, float z)
{
    x *= 3.0f;
    y *= 3.0f;
    z *= 3.0f;
    return (x*x*x*x - 5.0f*x*x +y*y*y*y - 5.0f*y*y +z*z*z*z - 5.0f*z*z + 11.8f) * 0.2f + 0.5f;
}

// evaluate field function at a point
// returns value and gradient in float4
__device__
float4 fieldFunc4(float3 p)
{
    float v = tangle(p.x, p.y, p.z);
    const float d = 0.001f;
    float dx = tangle(p.x + d, p.y, p.z) - v;
    float dy = tangle(p.x, p.y + d, p.z) - v;
    float dz = tangle(p.x, p.y, p.z + d) - v;
    return make_float4(dx, dy, dz, v);
}

__device__
void vertexInterp2(float isolevel, float3 p0, float3 p1, float4 f0, float4 f1, float3 &p, float3 &n)
{
    float t = (isolevel - f0.w) / (f1.w - f0.w);
    p = lerp(p0, p1, 0.5);
    n.x = lerp(f0.x, f1.x, t);
    n.y = lerp(f0.y, f1.y, t);
    n.z = lerp(f0.z, f1.z, t);
    n = normalize(n);
}

__device__
unsigned int getCell(float3 pos, BoundingBox boundingBox, unsigned int cubeSize)
{
    // "origin" of box is at the lower boundary
	float3 relPos = pos + -1.0 * boundingBox.lowerBoundary; 
    float sizeOfCell = (boundingBox.upperBoundary.x - boundingBox.lowerBoundary.x) / (float) cubeSize; 
    // Find which cell the position is in
    uint xCoord = (uint) floor(relPos.x / sizeOfCell); 
    uint yCoord = (uint) floor(relPos.y / sizeOfCell); 
    uint zCoord = (uint) floor(relPos.z / sizeOfCell); 
    return zCoord * cubeSize * cubeSize + yCoord * cubeSize + xCoord; 
}

__device__
bool isOutsideBoundingBox(float3 pos, BoundingBox BB)
{
    if (pos.x < BB.lowerBoundary.x || pos.y < BB.lowerBoundary.y || pos.z < BB.lowerBoundary.z) {
        return true; 
    }
    if (pos.x > BB.upperBoundary.x || pos.y > BB.upperBoundary.y || pos.z > BB.upperBoundary.z) {
        return true;
    }
    return false; 
}

__device__
BoundingBox calculateNewBoundingBox(float3 pos, BoundingBox boundingBox, uint cubeSize)
{
    // Find which cell of the old bounding box the pos is in
    float3 offsetFromOrigin = pos + (-1.0f * boundingBox.lowerBoundary);
    float sizeOfCell = (boundingBox.upperBoundary.x - boundingBox.lowerBoundary.x) / (float) cubeSize; 
    uint3 lowerIndex;
    lowerIndex.x = (uint) floor(offsetFromOrigin.x / sizeOfCell);
    lowerIndex.y = (uint) floor(offsetFromOrigin.y / sizeOfCell);
    lowerIndex.z = (uint) floor(offsetFromOrigin.z / sizeOfCell);
    // Calculate the new upper and lower boundaries based on the cell
    BoundingBox newBB; 
    newBB.lowerBoundary = boundingBox.lowerBoundary + make_float3(lowerIndex) * sizeOfCell; 
    newBB.upperBoundary = boundingBox.lowerBoundary + make_float3((lowerIndex + make_uint3(1,1,1))) * sizeOfCell; 
    if (isOutsideBoundingBox(pos, boundingBox)) {
        printf("Problem: Bounding box calculated incorrectly\n");
    }
    return newBB; 
}


__device__
unsigned int getStatus(float3 pos, float **pointersToStatuses, unsigned int **pointersToDelimiters)
{
    // Start at level 0, offset into cell 0, and the bounding box for the whole gdb
	unsigned int currentLevel = 0;
	BoundingBox currentBB = boundary;
	unsigned int offset = 0; 
    if (isOutsideBoundingBox(pos, currentBB)) {
        // If outside the bounding box, the voxel is inactive
        return 0.0; 
    }
	while (1) {
        // Otherwise, get the status of the cell we're in
		unsigned int cell = getCell(pos, currentBB, numCellsPerSide[currentLevel]);
		float status = pointersToStatuses[currentLevel][cell + offset];
		// Dig deeper = INF
		if (status != STATUS_FLAG_DIG_DEEPER) {
            // If it is active or inactive, return the status
			return status;
		} else {
            // Otherwise, find our new offset and bounding box, and loop
			unsigned int delimiter = pointersToDelimiters[currentLevel][cell + offset];
			unsigned int nextLevelCubeSize = numCellsPerSide[currentLevel + 1];
			offset = delimiter * nextLevelCubeSize * nextLevelCubeSize * nextLevelCubeSize; 
			currentBB = calculateNewBoundingBox(pos, currentBB, numCellsPerSide[currentLevel + 1]);
		}

	}
}

__global__
void calculateNewVelocities(float4 *particlePos,
                            float4 *particleVel,
                            float particleRadius,
                            unsigned int numParticles,
                            float **pointersToStatuses,
                            unsigned int **pointersToDelimiters,
                            float deltaTime, 
                            float4 *result,
                            unsigned int *sizeOfResult,
                            unsigned int maxResultSize)
{
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index >= numParticles) return;

    float3 currentParticlePos = make_float3(particlePos[index]);
    float3 currentParticleVel = make_float3(particleVel[index]);
    float iters = particleVel[index].w; 

    // Loop over all voxels that are touching the particle
    int loopStart = -1.0 * floor(particleRadius / voxelSize);
    int loopEnd = ceil(particleRadius / voxelSize); 

    float3 averagePosition = make_float3(0,0,0); 
    unsigned int numNeighboringVoxels = 0; 

    for (int z = loopStart; z <= loopEnd; ++z) {
    	for (int y = loopStart; y <= loopEnd; ++y) {
    		for (int x = loopStart; x <= loopEnd; ++x) {
    			float3 position = currentParticlePos + voxelSize * make_float3(x, y, z); 
    			float status = getStatus(position, pointersToStatuses, pointersToDelimiters); 
    			if (status > 0) {
                    // Get data for average voxel position
    				++numNeighboringVoxels; 
                    averagePosition += position;
                    // Reduce the strength (do so more for glancing blows) 
                    float t_c = 0.1; 
                    float amountToReduceStrength = -10.0 * length(cross(currentParticleVel, currentParticlePos - position)) * deltaTime / t_c;
                    unsigned int indexToAdd = atomicAdd(sizeOfResult, 1);
                    if (indexToAdd < maxResultSize) {
                        // Add position of voxel and amount to reduce strength to our output for later use
                        result[indexToAdd] = make_float4(position, amountToReduceStrength);  
                    }  
    			}
    		}
    	}
    }

    if (numNeighboringVoxels > 0) {
        // get the average position
        averagePosition.x = averagePosition.x / numNeighboringVoxels;
        averagePosition.y = averagePosition.y / numNeighboringVoxels;
        averagePosition.z = averagePosition.z / numNeighboringVoxels;
        
        // The particle reflects around the normal.  
        float3 normalVector = (currentParticlePos - averagePosition) / length(currentParticlePos - averagePosition);
        currentParticleVel -= 2 * dot(normalVector, currentParticleVel) * normalVector;
        currentParticlePos = averagePosition + (2 * particleRadius * normalVector);

        // TODO: Figure out a way to remove particles
    }
    particlePos[index] = make_float4(currentParticlePos, 1.0f);
    particleVel[index] = make_float4(currentParticleVel, iters);
}

__global__
void repairVoxelTree(const float4 *result,
                     float **pointersToStatuses,
                     unsigned int **pointersToDelimiters,
                     const unsigned int numToRepair,
                     unsigned int *numClaimedInArrayAtLevel,
                     unsigned int *addressOfErrorField)
{
    const uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index >= numToRepair) return;

    const float3 pos = make_float3(result[index]); 
    const float amountToReduceStrength = result[index].w;
    BoundingBox currentBB = boundary; 
    unsigned int cell; 
    unsigned int offset = 0;
    unsigned int numCellsInThisLevel = 1; 

    for (unsigned int level = 0; level < numLevels - 1; ++level) {
        // Get index of cell 
        cell = getCell(pos, currentBB, numCellsPerSide[level]);
        numCellsInThisLevel *= numCellsPerSide[level] *
          numCellsPerSide[level] * numCellsPerSide[level]; 
        if (cell + offset >= numCellsInThisLevel) {
            printf("Problem1 at index %d\n", index);
            atomicAdd(addressOfErrorField, unsigned(1));
            return; 
        }
        // First, check if the cell is DIG_DEEPER.  If it is, then we
        //  don't really need to do any fancy logic, we know we can just
        //  go ahead and dig deeper.
        const float firstStatusCheck =
          pointersToStatuses[level][cell + offset];
        if (firstStatusCheck != STATUS_FLAG_DIG_DEEPER) {
            // Now we know that it's either active or work in progress.
            // If it's active, we need to refine the cell.  If it's work
            //  in progress, then we know someone else is refining the cell
            //  and we just wait.
            // Check if work is happening already
            const float secondStatusCheck =
              atomicExch(&(pointersToStatuses[level][cell + offset]),
                         STATUS_FLAG_WORK_IN_PROGRESS);
            if (secondStatusCheck != STATUS_FLAG_WORK_IN_PROGRESS) {
                printf("Status seen by index %3u at level %3u, cell %3u offset %3u is %5f\n", index, level, cell, offset, secondStatusCheck);
                // If chunk is not allready dig deeper...
                const unsigned int chunkNumber =
                  pointersToDelimiters[level][cell + offset];
                /*printf("Index %5u is checking chunk number %5u\n",
                       index, chunkNumber);*/
                if (chunkNumber == INVALID_CHUNK_NUMBER) {
                    /*printf("Index %5u needs to refine cell on level %2u "
                           "offset %5u cell %3u\n",
                           index, level, offset, cell);*/
                    // Claim a chunk number at the next level
                    const unsigned int nextLevelsClaimedChunkNumber =
                      atomicAdd(&numClaimedInArrayAtLevel[level + 1], unsigned(1));
                    const unsigned int numCellsInChunkAtNextLevel =
                      numCellsPerSide[level + 1] * numCellsPerSide[level + 1] *
                      numCellsPerSide[level + 1];

                    const unsigned int numCellsInNextLevel =
                      numCellsInThisLevel * numCellsInChunkAtNextLevel;
                    for (unsigned int i = 0; i < numCellsInChunkAtNextLevel; ++i) {
                        if (nextLevelsClaimedChunkNumber * numCellsInChunkAtNextLevel + i >=
                            numCellsInNextLevel) {
                           printf("Index %5u is failing to set next chunk's "
                                  "values\n", index);
                           atomicAdd(addressOfErrorField, unsigned(1));
                           return; 
                        }
                        // Set next level to proper values
                        pointersToStatuses[level + 1][nextLevelsClaimedChunkNumber * numCellsInChunkAtNextLevel + i] = 1;
                        pointersToDelimiters[level + 1][nextLevelsClaimedChunkNumber * numCellsInChunkAtNextLevel + i] = INVALID_CHUNK_NUMBER; 
                    }
                    /*printf("Index %5u is setting offset of level %3u, cell %3u, offset %3u to %3u\n",
                       index, level, cell, offset, nextLevelsClaimedChunkNumber);*/
                    // Update chunk index
                    pointersToDelimiters[level][cell + offset] = nextLevelsClaimedChunkNumber; 
                }
                printf("Index %3u is setting status of level %3u, cell %3u, offset %3u at address %p\n",
                       index, level, cell, offset, &(pointersToStatuses[level][cell + offset]));
                pointersToStatuses[level][cell + offset] = STATUS_FLAG_DIG_DEEPER; 
            } else {
                unsigned int numberOfTimesWeveWaited = 0;
                while (pointersToStatuses[level][cell + offset] ==
                       STATUS_FLAG_WORK_IN_PROGRESS) {
                    ++numberOfTimesWeveWaited;
                    if (numberOfTimesWeveWaited > 9000000) {
                      printf("Index %2u is infinite looping "
                             "on cell %2u, offset %2u at level %2u, at address %p\n", index, cell, offset, level, &(pointersToStatuses[level][cell + offset]));
                      atomicAdd(addressOfErrorField, unsigned(1));
                      return;
                    }
                }
            }
        }
        const unsigned int delimiter = pointersToDelimiters[level][cell + offset];
        const unsigned int nextLevelCubeSize = numCellsPerSide[level + 1];
        offset = delimiter * nextLevelCubeSize * nextLevelCubeSize * nextLevelCubeSize; 
        currentBB = calculateNewBoundingBox(pos, currentBB, numCellsPerSide[level + 1]);
    }
    // TODO: Is amount to Reduce strength negative?
    atomicAdd(&pointersToStatuses[numLevels - 1][cell + offset], amountToReduceStrength);
    return; 	
}

__global__
void coarsenVoxelTree(float4 *result)
{
	return; 
}

void collideWithParticles(float *particlePos,
                          float *particleVel,
                          float  particleRadius,
                          unsigned int numParticles,
                          float **pointersToStatuses,
                          unsigned int **pointersToDelimiters,
                          unsigned int *numClaimedInArrayAtLevel,
                          float deltaTime)
{
	unsigned int numThreads = 256; 
	unsigned int numBlocks = ceil((float) numParticles / numThreads);
    unsigned int maxResultSize = 1000000;
	float *result;
    checkCudaErrors(hipMalloc((void **) &result, maxResultSize * sizeof(float4))); 
    unsigned int *sizeOfResult; 
    checkCudaErrors(hipMalloc((void **) &sizeOfResult, 1 * sizeof(unsigned int))); 
    checkCudaErrors(hipMemset(sizeOfResult, 0, sizeof(unsigned int))); 
    calculateNewVelocities<<<numBlocks, numThreads>>>((float4 *) particlePos,
                                                    (float4 *) particleVel,
                                                    particleRadius,
                                                    numParticles,
                                                    pointersToStatuses,
                                                    pointersToDelimiters,
                                                    deltaTime,
                                                    (float4 *) result,
                                                    sizeOfResult, 
                                                    maxResultSize);
    getLastCudaError("Kernel execution failed");


    unsigned int numberOfResultsProduced;
    hipMemcpy(&numberOfResultsProduced, sizeOfResult, sizeof(unsigned int),
               hipMemcpyDeviceToHost);
    numThreads = min(256, numberOfResultsProduced);
    if (numThreads > 0) {
        numBlocks = ceil((float) numberOfResultsProduced / numThreads);
        if (numberOfResultsProduced > maxResultSize) {
            fprintf(stderr, "problem: numberOfResultsProduced %d is greater "
                    "than max size %d\n",
                    numberOfResultsProduced, maxResultSize);
            exit(1);
        }
        const unsigned int numberOfResultsToProcess =
          std::min(numberOfResultsProduced, maxResultSize);
        unsigned int *addressOfErrorField;
        unsigned int zero = 0; 
        checkCudaErrors(hipMalloc((void **) &addressOfErrorField,
                                   1 * sizeof(unsigned int)));
        checkCudaErrors(hipMemcpy(addressOfErrorField, &zero,
                                   sizeof(unsigned int), hipMemcpyHostToDevice));
        if (numberOfResultsToProcess > 0) {
            printf("calling repairVoxelTree to process %4u results with "
                   "%4u threads and %4u blocks\n",
                   numberOfResultsToProcess, numThreads, numBlocks);
        }
        repairVoxelTree<<<numBlocks, numThreads>>>((float4 *) result,
                                                   pointersToStatuses,
                                                   pointersToDelimiters,
                                                   numberOfResultsToProcess,
                                                   numClaimedInArrayAtLevel,
                                                   addressOfErrorField);
        unsigned int numberOfErrors;
        hipMemcpy(&numberOfErrors, addressOfErrorField, sizeof(unsigned int),
                   hipMemcpyDeviceToHost);
        if (numberOfResultsToProcess > 0 || numberOfResultsToProcess > 0) {
            fprintf(stderr, "found %u errors after call to repairVoxelTree "
                    "with %u results\n", numberOfErrors,
                    numberOfResultsToProcess);
            exit(1);
        }
    }
    getLastCudaError("Kernel execution failed");

    //coarsenVoxelTree<<<numBlocks, numThreads>>>((float4 *) result);
    hipFree(result);
    hipFree(sizeOfResult); 
	
}

__global__
void createMarchingCubesMeshD(float4 *vertexPos,
                              float4 *norm,
                              float **pointersToStatuses,
                              unsigned int **pointersToDelimiters,
                              uint  *tri,
                              uint  *numVerts,
                              uint  *numVerticesClaimed,
                              uint   numVoxelsToDraw)
{
    uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

    // Get gridPos of our grid cube - starts as lower left corner as (0,0,0)
    uint3 gridPos = calculateCoordsFromIndex(index);
    //printf("GridPos: %d, %d, %d numCells: %d voxelSize: %f\n", gridPos.x, gridPos.y, gridPos.z, voxelsPerSide, voxelSize);
    
    // Check if voxels on corner of gridcube are 
    int lookupIndexForActiveVertices = 0;
    float3 cubeVertexPos[8];
    float4 field[8];

    int3 i = make_int3(-1, -1, -1);
    int3 toCheck = make_int3(gridPos) + i;
    cubeVertexPos[0] = calculateVoxelCenter(toCheck);
    bool isActive = getStatus(cubeVertexPos[0], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 0); 
    field[0] = fieldFunc4(cubeVertexPos[0]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 0, lookupIndexForActiveVertices, cubeVertexPos[0].x, cubeVertexPos[0].y, cubeVertexPos[0].z);

    
    i = make_int3(0,-1,-1);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[1] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[1], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 1); 
    field[1] = fieldFunc4(cubeVertexPos[1]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 1, lookupIndexForActiveVertices, cubeVertexPos[1].x, cubeVertexPos[1].y, cubeVertexPos[1].z);

    i = make_int3(0,0,-1);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[2] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[2], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 2); 
    field[2] = fieldFunc4(cubeVertexPos[2]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 2, lookupIndexForActiveVertices, cubeVertexPos[2].x, cubeVertexPos[2].y, cubeVertexPos[2].z);

    i = make_int3(-1,0,-1);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[3] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[3], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 3); 
    field[3] = fieldFunc4(cubeVertexPos[3]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 3, lookupIndexForActiveVertices, cubeVertexPos[3].x, cubeVertexPos[3].y, cubeVertexPos[3].z);

    i = make_int3(-1,-1,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[4] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[4], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 4); 
    field[4] = fieldFunc4(cubeVertexPos[4]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 4, lookupIndexForActiveVertices, cubeVertexPos[4].x, cubeVertexPos[4].y, cubeVertexPos[4].z);

    i = make_int3(0,-1,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[5] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[5], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 5); 
    field[5] = fieldFunc4(cubeVertexPos[5]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 5, lookupIndexForActiveVertices, cubeVertexPos[5].x, cubeVertexPos[5].y, cubeVertexPos[5].z);

    i = make_int3(0,0,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[6] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[6], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 6); 
    field[6] = fieldFunc4(cubeVertexPos[6]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 6, lookupIndexForActiveVertices, cubeVertexPos[6].x, cubeVertexPos[6].y, cubeVertexPos[6].z);

    i = make_int3(-1,0,0);
    toCheck = make_int3(gridPos) + i;
    cubeVertexPos[7] = calculateVoxelCenter(toCheck);
    isActive = getStatus(cubeVertexPos[7], pointersToStatuses, pointersToDelimiters) > 0;
    lookupIndexForActiveVertices = lookupIndexForActiveVertices | (isActive << 7); 
    field[7] = fieldFunc4(cubeVertexPos[7]);
    //printf("Num: %d Lookup: %d Pos: %f, %f, %f\n", 7, lookupIndexForActiveVertices, cubeVertexPos[7].x, cubeVertexPos[7].y, cubeVertexPos[7].z);
    
    float3 vertlist[12];
    float3 normList[12];

    vertexInterp2(0.0, cubeVertexPos[0], cubeVertexPos[1], field[0], field[1], vertlist[0], normList[0]);
    vertexInterp2(0.0, cubeVertexPos[1], cubeVertexPos[2], field[1], field[2], vertlist[1], normList[1]);
    vertexInterp2(0.0, cubeVertexPos[2], cubeVertexPos[3], field[2], field[3], vertlist[2], normList[2]);
    vertexInterp2(0.0, cubeVertexPos[3], cubeVertexPos[0], field[3], field[0], vertlist[3], normList[3]);

    vertexInterp2(0.0, cubeVertexPos[4], cubeVertexPos[5], field[4], field[5], vertlist[4], normList[4]);
    vertexInterp2(0.0, cubeVertexPos[5], cubeVertexPos[6], field[5], field[6], vertlist[5], normList[5]);
    vertexInterp2(0.0, cubeVertexPos[6], cubeVertexPos[7], field[6], field[7], vertlist[6], normList[6]);
    vertexInterp2(0.0, cubeVertexPos[7], cubeVertexPos[4], field[7], field[4], vertlist[7], normList[7]);

    vertexInterp2(0.0, cubeVertexPos[0], cubeVertexPos[4], field[0], field[4], vertlist[8], normList[8]);
    vertexInterp2(0.0, cubeVertexPos[1], cubeVertexPos[5], field[1], field[5], vertlist[9], normList[9]);
    vertexInterp2(0.0, cubeVertexPos[2], cubeVertexPos[6], field[2], field[6], vertlist[10], normList[10]);
    vertexInterp2(0.0, cubeVertexPos[3], cubeVertexPos[7], field[3], field[7], vertlist[11], normList[11]);

    uint numVerticesToAdd = tex1Dfetch(numVertsTex, lookupIndexForActiveVertices);
    uint positionToAdd = atomicAdd(numVerticesClaimed, numVerticesToAdd); 
    //if (gridPos.z ==0 && index % 1000 == 0) printf("To add: %d Pos: %d Total: %d\n", numVerticesToAdd, positionToAdd, numVoxelsToDraw * 15);
    for (int i= 0; i < numVerticesToAdd; ++i) {

        uint edge = tex1Dfetch(triTex, lookupIndexForActiveVertices*16 + i);
        uint indexToAdd = positionToAdd + i;

        if (indexToAdd < numVoxelsToDraw * 15)
        {
            vertexPos[indexToAdd] = make_float4(vertlist[edge], 1.0f);
            norm[indexToAdd] = make_float4(normList[edge], 0.0f);
        }
    }

}

void generateMarchingCubes(float *pos,
                           float *norm,
                           float **pointersToStatuses,
                           unsigned int **pointersToDelimiters,
                           unsigned int *tri,
                           unsigned int *numVerts,
                           unsigned int *verticesInPosArray,
                           unsigned int numVoxelsToDraw,
                           unsigned int numMarchingCubes)
{
    checkCudaErrors(hipBindTexture(0, triTex, tri, sizeof(uint) * 256 * 16));
    checkCudaErrors(hipBindTexture(0, numVertsTex, numVerts, sizeof(uint) * 256));

    // thread per particle
    uint numThreads, numBlocks;
    numThreads = 256;
    numBlocks = ceil((float) numMarchingCubes / (float) numThreads);
    hipMemset(verticesInPosArray, 0, sizeof(uint));

    // execute the kernel
    //timer->startTimer(5, true);
    createMarchingCubesMeshD<<< numBlocks, numThreads >>>((float4 *) pos,
                                                          (float4 *) norm,
                                                          pointersToStatuses,
                                                          pointersToDelimiters,
                                                           tri,
                                                           numVerts,
                                                           verticesInPosArray,
                                                           numVoxelsToDraw);
    //timer->stopTimer(5, true);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipUnbindTexture(triTex));
    checkCudaErrors(hipUnbindTexture(numVertsTex));
}
